#include "hip/hip_runtime.h"
/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <bits/stdint-uintn.h>
#include <include/cufhe.h>
#include <stdio.h>
#include <unistd.h>

#include <include/cufhe_gpu.cuh>
#include <include/bootstrap_gpu.cuh>
#include <include/details/error_gpu.cuh>
#include <include/ntt_gpu/ntt.cuh>
#include <iostream>
#include <limits>
#include <vector>

namespace cufhe {
using namespace std;
using namespace TFHEpp;

vector<FFP*> bk_ntts;
vector<CuNTTHandler<>*> ntt_handlers;
vector<lvl0param::T*> ksk_devs;

__global__ void __TRGSW2NTT__(FFP* bk_ntt, TFHEpp::lvl1param::T* bk,
                                          CuNTTHandler<> ntt)
{
    __shared__ FFP sh_temp[lvl1param::n];
    const int index = blockIdx.z * (2 * lvl1param::l * 2 * lvl1param::n) +
                      blockIdx.y * 2 * lvl1param::n + blockIdx.x * lvl1param::n;
    ntt.NTT<lvl1param::T>(&bk_ntt[index], &bk[index], sh_temp, 0);
}

void TRGSW2NTT(cuFHETRGSWNTTlvl1& trgswntt, const TFHEpp::TRGSW<TFHEpp::lvl1param>& trgsw, Stream st){
    hipSetDevice(st.device_id());
    TFHEpp::lvl1param::T* d_trgsw;
    hipMalloc((void**)&d_trgsw, sizeof(trgsw));
    hipMemcpyAsync(d_trgsw, trgsw.data(), sizeof(trgsw), hipMemcpyHostToDevice,st.st());

    dim3 grid(2, 2 * lvl1param::l, 1);
    dim3 block(lvl1param::n >> NTT_THRED_UNITBIT);
    __TRGSW2NTT__<<<grid, block, 0, st.st()>>>(trgswntt.trgswdevices[st.device_id()], d_trgsw,
                                                *ntt_handlers[st.device_id()]);
    CuCheckError();
    hipMemcpyAsync(trgswntt.trgswhost.data(), trgswntt.trgswdevices[st.device_id()],
                    sizeof(trgswntt.trgswhost), hipMemcpyDeviceToHost, st.st());
    hipFree(d_trgsw);
}

void InitializeNTThandlers(const int gpuNum){
    for (int i = 0; i < gpuNum; i++) {
        hipSetDevice(i);

        ntt_handlers.push_back(new CuNTTHandler<>());
        ntt_handlers[i]->Create();
        ntt_handlers[i]->CreateConstant();
        hipDeviceSynchronize();
        CuCheckError();
    }
}

void BootstrappingKeyToNTT(const BootstrappingKey<lvl01param>& bk,
                           const int gpuNum)
{
    bk_ntts.resize(gpuNum);
    for (int i = 0; i < gpuNum; i++) {
        hipSetDevice(i);

        hipMalloc((void**)&bk_ntts[i], sizeof(FFP) * lvl0param::n * 2 *
                                            lvl1param::l * 2 * lvl1param::n);

        TFHEpp::lvl1param::T* d_bk;
        hipMalloc((void**)&d_bk, sizeof(bk));
        hipMemcpy(d_bk, bk.data(), sizeof(bk), hipMemcpyHostToDevice);

        hipDeviceSynchronize();
        CuCheckError();

        dim3 grid(2, 2 * lvl1param::l, lvl0param::n);
        dim3 block(lvl1param::n >> NTT_THRED_UNITBIT);
        __TRGSW2NTT__<<<grid, block>>>(bk_ntts[i], d_bk,
                                                   *ntt_handlers[i]);
        hipDeviceSynchronize();
        CuCheckError();

        hipFree(d_bk);
    }
}

void DeleteBootstrappingKeyNTT(const int gpuNum)
{
    for (int i = 0; i < bk_ntts.size(); i++) {
        hipSetDevice(i);
        hipFree(bk_ntts[i]);

        ntt_handlers[i]->Destroy();
        delete ntt_handlers[i];
    }
    ntt_handlers.clear();
}

void KeySwitchingKeyToDevice(const KeySwitchingKey<lvl10param>& ksk,
                             const int gpuNum)
{
    ksk_devs.resize(gpuNum);
    for (int i = 0; i < gpuNum; i++) {
        hipSetDevice(i);
        hipMalloc((void**)&ksk_devs[i], sizeof(ksk));
        CuSafeCall(hipMemcpy(ksk_devs[i], ksk.data(), sizeof(ksk),
                              hipMemcpyHostToDevice));
    }
}

void DeleteKeySwitchingKey(const int gpuNum)
{
    for (int i = 0; i < ksk_devs.size(); i++) {
        hipSetDevice(i);
        hipFree(ksk_devs[i]);
    }
}

template <class P>
__device__ inline typename P::T modSwitchFromTorus(const uint32_t phase)
{
    constexpr uint32_t Mbit = P::nbit + 1;
    static_assert(32 >= Mbit, "Undefined modSwitchFromTorus!");
    return (phase + (1U << (31 - Mbit))) >> (32 - Mbit);
}

template <class P>
__device__ inline void KeySwitch(typename P::targetP::T* lwe,
                                 const typename P::domainP::T* const tlwe,
                                 const typename P::targetP::T* const ksk)
{
    constexpr typename P::domainP::T decomp_mask = (1U << P::basebit) - 1;
    constexpr typename P::domainP::T decomp_offset =
        1U << (std::numeric_limits<typename P::domainP::T>::digits - 1 -
               P::t * P::basebit);
    const uint32_t tid = ThisThreadRankInBlock();
    const uint32_t bdim = ThisBlockSize();
    for (int i = tid; i <= P::targetP::n; i += bdim) {
        typename P::targetP::T res = 0;
        if (i == P::targetP::n) res = tlwe[P::domainP::n];
        for (int j = 0; j < P::domainP::n; j++) {
            typename P::domainP::T tmp;
            if (j == 0)
                tmp = tlwe[0];
            else
                tmp = -tlwe[P::domainP::n - j];
            tmp += decomp_offset;
            for (int k = 0; k < P::t; k++) {
                typename P::domainP::T val =
                    (tmp >>
                     (std::numeric_limits<typename P::domainP::T>::digits -
                      (k + 1) * P::basebit)) &
                    decomp_mask;
                if (val != 0) {
                    constexpr int numbase = (1 << P::basebit) - 1;
                    res -= ksk[j * (lvl10param::t * numbase *
                                    (P::targetP::n + 1)) +
                               k * (numbase * (P::targetP::n + 1)) +
                               (val - 1) * (P::targetP::n + 1) + i];
                }
            }
        }
        lwe[i] = res;
    }
}

template <class P>
__device__ constexpr typename P::T offsetgen()
{
    typename P::T offset = 0;
    for (int i = 1; i <= P::l; i++)
        offset +=
            P::Bg / 2 *
            (1ULL << (numeric_limits<typename P::T>::digits - i * P::Bgbit));
    return offset;
}

__device__ inline void TRLWESubAndDecomposition(
    FFP* dectrlwe, const TFHEpp::lvl1param::T* const trlwe1, const TFHEpp::lvl1param::T* const trlwe0)
{
    const uint32_t tid = ThisThreadRankInBlock();
    const uint32_t bdim = ThisBlockSize();
    constexpr uint32_t decomp_mask = (1 << lvl1param::Bgbit) - 1;
    constexpr int32_t decomp_half = 1 << (lvl1param::Bgbit - 1);
    constexpr uint32_t decomp_offset = offsetgen<lvl1param>();
    constexpr typename lvl1param::T roundoffset = 1ULL<<(std::numeric_limits<typename lvl1param::T>::digits-lvl1param::l*lvl1param::Bgbit-1);
#pragma unroll
    for (int i = tid; i < lvl1param::n; i += bdim) {
        #pragma unroll
        for(int j = 0; j < 2; j ++){
            // decomp temp
            lvl1param::T temp = trlwe1[j*lvl1param::n+i]-trlwe0[j*lvl1param::n+i]+decomp_offset+roundoffset;
    #pragma unroll
            for (int digit = 0; digit < lvl1param::l; digit += 1)
                dectrlwe[j*lvl1param::l*lvl1param::n+digit * lvl1param::n + i] = FFP(lvl1param::T(
                    ((temp >> (std::numeric_limits<typename lvl1param::T>::digits -
                            (digit + 1) * lvl1param::Bgbit)) &
                    decomp_mask) -
                    decomp_half));
        }
    }
    __syncthreads();  // must
}

__global__ void __CMUXNTT__(TFHEpp::lvl1param::T* out, const FFP* const tgsw_ntt,
                                  const TFHEpp::lvl1param::T* const trlwe1,
                                  const TFHEpp::lvl1param::T* const trlwe0,
                                  const CuNTTHandler<> ntt)
{
    const uint32_t tid = ThisThreadRankInBlock();
    const uint32_t bdim = ThisBlockSize();

    extern __shared__ FFP sh[];
    // To hold the data after Decomposition and NTT
    FFP* sh_acc_ntt = &sh[0];
    // To hold sum
    FFP* sh_res_ntt = &sh[2*lvl1param::l*lvl1param::n];
    TFHEpp::lvl1param::T* outtemp =
        (TFHEpp::lvl1param::T*)&sh[0];

    TRLWESubAndDecomposition(sh_acc_ntt, trlwe1, trlwe0);

    // 2*l NTTs
    // Input/output/buffer use the same shared memory location.
    if (tid < 2*lvl1param::l * (lvl1param::n >> NTT_THRED_UNITBIT)) {
        FFP* tar = &sh_acc_ntt[tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                                       << lvl1param::nbit];
        ntt.NTT<FFP>(tar, tar, tar,
                     tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                                << (lvl1param::nbit - NTT_THRED_UNITBIT));
    }
    else {  // must meet 5 sync made by NTTInv
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
    }
    __syncthreads();

// Multiply with bootstrapping key in global memory.
#pragma unroll
    for (int i = tid; i < lvl1param::n; i += bdim) {
        sh_res_ntt[i] = sh_acc_ntt[0 * lvl1param::n + i] *
                        tgsw_ntt[((2 * 0 + 0) << lvl1param::nbit) + i];
        sh_res_ntt[i + lvl1param::n] =
            sh_acc_ntt[0 * lvl1param::n + i] *
            tgsw_ntt[((2 * 0 + 1) << lvl1param::nbit) + i];
#pragma unroll
        for (int digit = 1; digit < 2*lvl1param::l; digit += 1) {
            sh_res_ntt[i] += sh_acc_ntt[digit * lvl1param::n + i] *
                             tgsw_ntt[((2 * digit + 0) << lvl1param::nbit) + i];
            sh_res_ntt[i + lvl1param::n] +=
                sh_acc_ntt[digit * lvl1param::n + i] *
                tgsw_ntt[((2 * digit + 1) << lvl1param::nbit) + i];
        }
    }
    __syncthreads();

    #pragma unroll
    for (int i = tid; i < 2*lvl1param::n; i += bdim) outtemp[i] = trlwe0[i];

    // 2 NTTInvs and add acc
    if (tid < 2 * (lvl1param::n >> NTT_THRED_UNITBIT)) {
        FFP* src = &sh_res_ntt[tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                                          << lvl1param::nbit];
        ntt.NTTInvAdd<typename lvl1param::T>(
            &outtemp[tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                             << lvl1param::nbit],
            src, src,
            tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                       << (lvl1param::nbit - NTT_THRED_UNITBIT));
    }
    else {  // must meet 5 sync made by NTTInv
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
    }
    __syncthreads();  // must
    for(int i = 0; i<2*lvl1param::n;i++) out[i] = outtemp[i];
    __syncthreads();
}

template <class P>
__device__ inline void RotatedTestVector(TFHEpp::lvl1param::T* tlwe,
                                         const int32_t bar,
                                         const typename P::T μ)
{
    // volatile is needed to make register usage of Mux to 128.
    // Reference
    // https://devtalk.nvidia.com/default/topic/466758/cuda-programming-and-performance/tricks-to-fight-register-pressure-or-how-i-got-down-from-29-to-15-registers-/
    volatile const uint32_t tid = ThisThreadRankInBlock();
    volatile const uint32_t bdim = ThisBlockSize();
#pragma unroll
    for (int i = tid; i < P::n; i += bdim) {
        tlwe[i] = 0;  // part a
        if (bar == 2 * P::n)
            tlwe[i + P::n] = μ;
        else {
            tlwe[i + P::n] = ((i < (bar & (P::n - 1))) ^ (bar >> P::nbit))
                                 ? -μ
                                 : μ;  // part b
        }
    }
    __syncthreads();
}

__device__ inline void PolynomialMulByXaiMinusOneAndDecompositionTRLWE(
    FFP* dectrlwe, const TFHEpp::lvl1param::T* const trlwe, const uint32_t a_bar)
{
    const uint32_t tid = ThisThreadRankInBlock();
    const uint32_t bdim = ThisBlockSize();
    constexpr uint32_t decomp_mask = (1 << lvl1param::Bgbit) - 1;
    constexpr int32_t decomp_half = 1 << (lvl1param::Bgbit - 1);
    constexpr uint32_t decomp_offset = offsetgen<lvl1param>();
    constexpr typename lvl1param::T roundoffset = 1ULL<<(std::numeric_limits<typename lvl1param::T>::digits-lvl1param::l*lvl1param::Bgbit-1);
#pragma unroll
    for (int i = tid; i < lvl1param::n; i += bdim) {
        #pragma unroll
        for (int j = 0; j < 2; j++){
            //PolynomialMulByXaiMinus
            lvl1param::T temp = trlwe[j*lvl1param::n + ((i - a_bar) & (lvl1param::n - 1))];
            temp = ((i < (a_bar & (lvl1param::n - 1)) ^ (a_bar >> lvl1param::nbit)))
                    ? -temp
                    : temp;
            temp -= trlwe[j*lvl1param::n + i];
            // decomp temp
            temp += decomp_offset + roundoffset;
    #pragma unroll
            for (int digit = 0; digit < lvl1param::l; digit += 1)
                dectrlwe[j*lvl1param::l*lvl1param::n + digit * lvl1param::n + i] = FFP(lvl1param::T(
                    ((temp >> (std::numeric_limits<typename lvl1param::T>::digits -
                            (digit + 1) * lvl1param::Bgbit)) &
                    decomp_mask) -
                    decomp_half));
        }
    }
    __syncthreads();  // must
}

__device__ inline void Accumulate(TFHEpp::lvl1param::T* trlwe,
                                  FFP* sh_acc_ntt, const uint32_t a_bar,
                                  const FFP* const tgsw_ntt,
                                  const CuNTTHandler<> ntt)
{
    const uint32_t tid = ThisThreadRankInBlock();
    const uint32_t bdim = ThisBlockSize();

    PolynomialMulByXaiMinusOneAndDecompositionTRLWE(sh_acc_ntt, trlwe, a_bar);

    // 2l NTTs
    // Input/output/buffer use the same shared memory location.
    if (tid < 2*lvl1param::l * (lvl1param::n >> NTT_THRED_UNITBIT)) {
        FFP* tar = &sh_acc_ntt[tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                                       << lvl1param::nbit];
        ntt.NTT<FFP>(tar, tar, tar,
                     tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                                << (lvl1param::nbit - NTT_THRED_UNITBIT));
    }
    else {  // must meet 5 sync made by NTTInv
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
    }
    __syncthreads();

// Multiply with bootstrapping key in global memory.
#pragma unroll
    for (int i = tid; i < lvl1param::n; i += bdim) {
        sh_acc_ntt[2*lvl1param::l*lvl1param::n + i] =
            sh_acc_ntt[0 * lvl1param::n + i] *
            tgsw_ntt[((2 * 0 + 1) << lvl1param::nbit) + i];
        sh_acc_ntt[i] = sh_acc_ntt[0 * lvl1param::n + i] *
                        tgsw_ntt[((2 * 0 + 0) << lvl1param::nbit) + i];
#pragma unroll
        for (int digit = 1; digit < 2*lvl1param::l; digit += 1) {
            sh_acc_ntt[i] += sh_acc_ntt[digit * lvl1param::n + i] *
                             tgsw_ntt[((2 * digit + 0) << lvl1param::nbit) + i];
            sh_acc_ntt[2*lvl1param::l*lvl1param::n + i] +=
                sh_acc_ntt[digit * lvl1param::n + i] *
                tgsw_ntt[((2 * digit + 1) << lvl1param::nbit) + i];
        }
    }
    __syncthreads();

    // 2 NTTInvs and add acc
    if (tid < 2 * (lvl1param::n >> NTT_THRED_UNITBIT)) {
        FFP* src = &sh_acc_ntt[(tid >> (lvl1param::nbit - NTT_THRED_UNITBIT))*2*lvl1param::l*lvl1param::n];
        ntt.NTTInvAdd<typename lvl1param::T>(
            &trlwe[tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                             << lvl1param::nbit],
            src, src,
            tid >> (lvl1param::nbit - NTT_THRED_UNITBIT)
                       << (lvl1param::nbit - NTT_THRED_UNITBIT));
    }
    else {  // must meet 5 sync made by NTTInv
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
        __syncthreads();
    }
    __syncthreads();  // must
}

__global__ void __Bootstrap__(TFHEpp::lvl0param::T* out,
                              TFHEpp::lvl0param::T* in,
                              const TFHEpp::lvl1param::T mu,
                              const FFP* const bk,
                              const TFHEpp::lvl0param::T* const ksk,
                              const CuNTTHandler<> ntt)
{
    //  Assert(bk.k() == 1);
    //  Assert(bk.l() == 2);
    //  Assert(bk.n() == lvl1param::n);
    extern __shared__ FFP sh[];
    FFP* sh_acc_ntt = &sh[0];
    // Use Last section to hold tlwe. This may to make these data in serial
    TFHEpp::lvl1param::T* tlwe =
        (TFHEpp::lvl1param::T*)&sh[(2 * lvl1param::l + 1) * lvl1param::n];

    // test vector
    // acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    {
        const uint32_t bar =
            2 * lvl1param::n - modSwitchFromTorus<lvl1param>(in[lvl0param::n]);
        RotatedTestVector<lvl1param>(tlwe, bar, mu);
    }

    // accumulate
    for (int i = 0; i < lvl0param::n; i++) {  // n iterations
        const uint32_t bar = modSwitchFromTorus<lvl1param>(in[i]);
        Accumulate(tlwe, sh_acc_ntt, bar,
                   bk + (i << lvl1param::nbit) * 2 * 2 * lvl1param::l, ntt);
    }

    KeySwitch<lvl10param>(out, tlwe, ksk);
    __threadfence();
}

__global__ void __SEandKS__(TFHEpp::lvl0param::T* out, TFHEpp::lvl1param::T* in,
                            FFP* bk, TFHEpp::lvl0param::T* ksk)
{
    KeySwitch<lvl10param>(out, in, ksk);
    __threadfence();
}

__global__ void __BootstrapTLWE2TRLWE__(TFHEpp::lvl1param::T* out,
                                        TFHEpp::lvl0param::T* in,
                                        TFHEpp::lvl1param::T mu, FFP* bk,
                                        CuNTTHandler<> ntt)
{
    //  Assert(bk.k() == 1);
    //  Assert(bk.l() == 2);
    //  Assert(bk.n() == lvl1param::n);
    extern __shared__ FFP sh[];
    FFP* sh_acc_ntt = &sh[0];
    // Use Last section to hold tlwe. This may to make these data in serial
    TFHEpp::lvl1param::T* tlwe =
        (TFHEpp::lvl1param::T*)&sh[(2 * lvl1param::l + 1) * lvl1param::n];

    // test vector
    // acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register uint32_t bar =
        2 * lvl1param::n - modSwitchFromTorus<lvl1param>(in[lvl0param::n]);
    RotatedTestVector<lvl1param>(tlwe, bar, mu);

    // accumulate
    for (int i = 0; i < lvl0param::n; i++) {  // n iterations
        bar = modSwitchFromTorus<lvl1param>(in[i]);
        Accumulate(tlwe, sh_acc_ntt, bar,
                   bk + (i << lvl1param::nbit) * 2 * 2 * lvl1param::l, ntt);
    }
    __syncthreads();
    for (int i = 0; i < 2 * lvl1param::n; i++) {
        out[i] = tlwe[i];
    }
    __threadfence();
}

__global__ void __SEIandBootstrap2TRLWE__(TFHEpp::lvl1param::T* out,
                                        TFHEpp::lvl1param::T* in,
                                        TFHEpp::lvl1param::T mu, FFP* bk,
                                        TFHEpp::lvl0param::T* ksk,
                                        CuNTTHandler<> ntt)
{
    //  Assert(bk.k() == 1);
    //  Assert(bk.l() == 2);
    //  Assert(bk.n() == lvl1param::n);
    extern __shared__ FFP sh[];
    FFP* sh_acc_ntt = &sh[0];
    // Use Last section to hold tlwe. This may to make these data in serial
    TFHEpp::lvl1param::T* tlwe =
        (TFHEpp::lvl1param::T*)&sh[(2 * lvl1param::l + 2) * lvl1param::n];

    lvl0param::T* tlwelvl0 = (lvl0param::T*)&sh[(2 * lvl1param::l + 3) * lvl1param::n];

    KeySwitch<lvl10param>(tlwelvl0, in, ksk);
    __syncthreads();

    // test vector
    // acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register uint32_t bar =
        2 * lvl1param::n - modSwitchFromTorus<lvl1param>(tlwelvl0[lvl0param::n]);
    RotatedTestVector<lvl1param>(tlwe, bar, mu);

    // accumulate
    for (int i = 0; i < lvl0param::n; i++) {  // n iterations
        bar = modSwitchFromTorus<lvl1param>(tlwelvl0[i]);
        Accumulate(tlwe, sh_acc_ntt, bar,
                   bk + (i << lvl1param::nbit) * 2 * 2 * lvl1param::l, ntt);
    }
    __syncthreads();
    for (int i = 0; i < 2 * lvl1param::n; i++) {
        out[i] = tlwe[i];
    }
    __threadfence();
}

template <int casign, int cbsign, typename lvl0param::T offset>
__device__ inline void __HomGate__(TFHEpp::lvl0param::T* out,
                                   TFHEpp::lvl0param::T* in0,
                                   TFHEpp::lvl0param::T* in1, FFP* bk,
                                   TFHEpp::lvl0param::T* ksk,
                                   CuNTTHandler<> ntt)
{
    extern __shared__ FFP sh[];
    FFP* sh_acc_ntt = &sh[0];
    // Use Last section to hold tlwe. This may to make these data in serial
    TFHEpp::lvl1param::T* tlwe =
        (TFHEpp::lvl1param::T*)&sh[(2 * lvl1param::l + 1) * lvl1param::n];

    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    {
        const uint32_t bar =
            2 * lvl1param::n -
            modSwitchFromTorus<lvl1param>(offset + casign * in0[lvl0param::n] +
                                          cbsign * in1[lvl0param::n]);
        RotatedTestVector<lvl1param>(tlwe, bar, lvl1param::μ);
    }

    // accumulate
    for (int i = 0; i < lvl0param::n; i++) {  // lvl0param::n iterations
        const uint32_t bar = modSwitchFromTorus<lvl1param>(0 + casign * in0[i] +
                                                           cbsign * in1[i]);
        Accumulate(tlwe, sh_acc_ntt, bar,
                   bk + (i << lvl1param::nbit) * 2 * 2 * lvl1param::l, ntt);
    }
    KeySwitch<lvl10param>(out, tlwe, ksk);
    __threadfence();
}

__global__ void __NandBootstrap__(TFHEpp::lvl0param::T* out,
                                  TFHEpp::lvl0param::T* in0,
                                  TFHEpp::lvl0param::T* in1, FFP* bk,
                                  TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    __HomGate__<-1, -1, lvl0param::μ>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __NorBootstrap__(TFHEpp::lvl0param::T* out,
                                 TFHEpp::lvl0param::T* in0,
                                 TFHEpp::lvl0param::T* in1, FFP* bk,
                                 TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    __HomGate__<-1, -1, -lvl0param::μ>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __XnorBootstrap__(TFHEpp::lvl0param::T* out,
                                  TFHEpp::lvl0param::T* in0,
                                  TFHEpp::lvl0param::T* in1, FFP* bk,
                                  TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    __HomGate__<-2, -2, -2 * lvl0param::μ>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __AndBootstrap__(TFHEpp::lvl0param::T* out,
                                 TFHEpp::lvl0param::T* in0,
                                 TFHEpp::lvl0param::T* in1, FFP* bk,
                                 TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    __HomGate__<1, 1, -lvl0param::μ>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __OrBootstrap__(TFHEpp::lvl0param::T* out,
                                TFHEpp::lvl0param::T* in0,
                                TFHEpp::lvl0param::T* in1, FFP* bk,
                                TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    __HomGate__<1, 1, lvl0param::μ>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __XorBootstrap__(TFHEpp::lvl0param::T* out,
                                 TFHEpp::lvl0param::T* in0,
                                 TFHEpp::lvl0param::T* in1, FFP* bk,
                                 TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    __HomGate__<2, 2, 2*lvl0param::μ>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __AndNYBootstrap__(TFHEpp::lvl0param::T* out,
                                   TFHEpp::lvl0param::T* in0,
                                   TFHEpp::lvl0param::T* in1, FFP* bk,
                                   TFHEpp::lvl0param::T* ksk,
                                   CuNTTHandler<> ntt)
{
    __HomGate__<-1, 1, -lvl0param::μ>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __AndYNBootstrap__(TFHEpp::lvl0param::T* out,
                                   TFHEpp::lvl0param::T* in0,
                                   TFHEpp::lvl0param::T* in1, FFP* bk,
                                   TFHEpp::lvl0param::T* ksk,
                                   CuNTTHandler<> ntt)
{
    __HomGate__<1, -1, -lvl0param::μ>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __OrNYBootstrap__(TFHEpp::lvl0param::T* out,
                                  TFHEpp::lvl0param::T* in0,
                                  TFHEpp::lvl0param::T* in1, FFP* bk,
                                  TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    __HomGate__<-1, 1, lvl0param::μ>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __OrYNBootstrap__(TFHEpp::lvl0param::T* out,
                                  TFHEpp::lvl0param::T* in0,
                                  TFHEpp::lvl0param::T* in1, FFP* bk,
                                  TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    __HomGate__<1, -1, lvl0param::μ>(out, in0, in1, bk, ksk, ntt);
}

__global__ void __CopyBootstrap__(TFHEpp::lvl0param::T* out,
                                  TFHEpp::lvl0param::T* in)
{
    const uint32_t tid = ThisThreadRankInBlock();
    out[tid] = in[tid];
    __syncthreads();
    __threadfence();
}

__global__ void __NotBootstrap__(TFHEpp::lvl0param::T* out,
                                 TFHEpp::lvl0param::T* in)
{
    const uint32_t tid = ThisThreadRankInBlock();
    out[tid] = -in[tid];
    __syncthreads();
    __threadfence();
}

// Mux(inc,in1,in0) = inc?in1:in0 = inc&in1 + (!inc)&in0
__global__ void __MuxBootstrap__(TFHEpp::lvl0param::T* out,
                                 TFHEpp::lvl0param::T* inc,
                                 TFHEpp::lvl0param::T* in1,
                                 TFHEpp::lvl0param::T* in0, FFP* bk,
                                 TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    // To use over 48 KiB shared Memory, the dynamic allocation is required.
    extern __shared__ FFP sh[];
    FFP* sh_acc_ntt = &sh[0];
    // Use Last section to hold tlwe. This may to make these data in serial
    TFHEpp::lvl1param::T* tlwe1 =
        (TFHEpp::lvl1param::T*)&sh[(2*lvl1param::l + 1) * lvl1param::n];
    TFHEpp::lvl1param::T* tlwe0 =
        (TFHEpp::lvl1param::T*)&sh[(2*lvl1param::l + 2) * lvl1param::n];
    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register uint32_t bar =
        2 * lvl1param::n -
        modSwitchFromTorus<lvl1param>(-lvl0param::μ + inc[lvl0param::n] +
                                      in1[lvl0param::n]);
    RotatedTestVector<lvl1param>(tlwe1, bar, lvl1param::μ);

    // accumulate
    for (int i = 0; i < lvl0param::n; i++) {  // lvl1param::n iterations
        bar = modSwitchFromTorus<lvl1param>(0 + inc[i] + in1[i]);
        Accumulate(tlwe1, sh_acc_ntt, bar,
                   bk + (i << lvl1param::nbit) * 2 * 2 * lvl1param::l, ntt);
    }

    bar = 2 * lvl1param::n -
          modSwitchFromTorus<lvl1param>(-lvl0param::μ - inc[lvl0param::n] +
                                        in0[lvl0param::n]);

    RotatedTestVector<lvl1param>(tlwe0, bar, lvl1param::μ);

    for (int i = 0; i < lvl0param::n; i++) {  // lvl1param::n iterations
        bar = modSwitchFromTorus<lvl1param>(0 - inc[i] + in0[i]);
        Accumulate(tlwe0, sh_acc_ntt, bar,
                   bk + (i << lvl1param::nbit) * 2 * 2 * lvl1param::l, ntt);
    }

    volatile const uint32_t tid = ThisThreadRankInBlock();
    volatile const uint32_t bdim = ThisBlockSize();
#pragma unroll
    for (int i = tid; i <= lvl1param::n; i += bdim) {
        tlwe1[i] += tlwe0[i];
        if (i == lvl1param::n) {
            tlwe1[lvl1param::n] += lvl1param::μ;
        }
    }

    __syncthreads();

    KeySwitch<lvl10param>(out, tlwe1, ksk);
    __threadfence();
}

// NMux(inc,in1,in0) = !(inc?in1:in0) = !(inc&in1 + (!inc)&in0)
__global__ void __NMuxBootstrap__(TFHEpp::lvl0param::T* out,
                                 TFHEpp::lvl0param::T* inc,
                                 TFHEpp::lvl0param::T* in1,
                                 TFHEpp::lvl0param::T* in0, FFP* bk,
                                 TFHEpp::lvl0param::T* ksk, CuNTTHandler<> ntt)
{
    // To use over 48 KiB shared Memory, the dynamic allocation is required.
    extern __shared__ FFP sh[];
    FFP* sh_acc_ntt = &sh[0];
    // Use Last section to hold tlwe. This may to make these data in serial
    TFHEpp::lvl1param::T* tlwe1 =
        (TFHEpp::lvl1param::T*)&sh[(2*lvl1param::l + 1) * lvl1param::n];
    TFHEpp::lvl1param::T* tlwe0 =
        (TFHEpp::lvl1param::T*)&sh[(2*lvl1param::l + 2) * lvl1param::n];
    
    // test vector: acc.a = 0; acc.b = vec(mu) * x ^ (in.b()/2048)
    register uint32_t bar =
        2 * lvl1param::n -
        modSwitchFromTorus<lvl1param>(-lvl0param::μ + inc[lvl0param::n] +
                                      in1[lvl0param::n]);
    RotatedTestVector<lvl1param>(tlwe1, bar, lvl1param::μ);

    // accumulate
    for (int i = 0; i < lvl0param::n; i++) {  // lvl1param::n iterations
        bar = modSwitchFromTorus<lvl1param>(0 + inc[i] + in1[i]);
        Accumulate(tlwe1, sh_acc_ntt, bar,
                   bk + (i << lvl1param::nbit) * 2 * 2 * lvl1param::l, ntt);
    }

    bar = 2 * lvl1param::n -
          modSwitchFromTorus<lvl1param>(-lvl0param::μ - inc[lvl0param::n] +
                                        in0[lvl0param::n]);

    RotatedTestVector<lvl1param>(tlwe0, bar, lvl1param::μ);

    for (int i = 0; i < lvl0param::n; i++) {  // lvl1param::n iterations
        bar = modSwitchFromTorus<lvl1param>(0 - inc[i] + in0[i]);
        Accumulate(tlwe0, sh_acc_ntt, bar,
                   bk + (i << lvl1param::nbit) * 2 * 2 * lvl1param::l, ntt);
    }

    volatile const uint32_t tid = ThisThreadRankInBlock();
    volatile const uint32_t bdim = ThisBlockSize();
#pragma unroll
    for (int i = tid; i <= lvl1param::n; i += bdim) {
        tlwe1[i] = -tlwe1[i]-tlwe0[i];
        if (i == lvl1param::n) {
            tlwe1[lvl1param::n] -= lvl1param::μ;
        }
    }

    __syncthreads();

    KeySwitch<lvl10param>(out, tlwe1, ksk);
    __threadfence();
}

constexpr uint NUM_THREAD4HOMGATE = 2 * lvl1param::l * lvl1param::n >>  NTT_THRED_UNITBIT;

void Bootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in,
               lvl1param::T mu, hipStream_t st, const int gpuNum)
{
    __Bootstrap__<<<1, NUM_THREAD4HOMGATE, 0,
                  st>>>
        (out, in, mu, bk_ntts[gpuNum], ksk_devs[gpuNum], *ntt_handlers[gpuNum]);
    CuCheckError();
}

void SEandKS(TFHEpp::lvl0param::T* out, TFHEpp::lvl1param::T* in,
             hipStream_t st, const int gpuNum)
{
    __SEandKS__<<<1, lvl0param::n + 1, 0, st>>>(out, in, bk_ntts[gpuNum],
                                                ksk_devs[gpuNum]);
    CuCheckError();
}

void CMUXNTTkernel(TFHEpp::lvl1param::T* res, const FFP* const cs, TFHEpp::lvl1param::T* const c1, TFHEpp::lvl1param::T* const c0,
                         hipStream_t st, const int gpuNum)
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        __CMUXNTT__), hipFuncAttributeMaxDynamicSharedMemorySize,
        (2*lvl1param::l + 2) * lvl1param::n * sizeof(FFP));
    __CMUXNTT__<<<1, NUM_THREAD4HOMGATE,
                            (2*lvl1param::l + 2) * lvl1param::n * sizeof(FFP), st>>>
        (res, cs, c1, c0, *ntt_handlers[gpuNum]);
    CuCheckError();
}

constexpr uint MEM4HOMGATE = (2*lvl1param::l + 2) * lvl1param::n * sizeof(FFP);

void BootstrapTLWE2TRLWE(TFHEpp::lvl1param::T* out, TFHEpp::lvl0param::T* in,
                         lvl1param::T mu, hipStream_t st, const int gpuNum)
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        __BootstrapTLWE2TRLWE__), hipFuncAttributeMaxDynamicSharedMemorySize,
        MEM4HOMGATE);
    __BootstrapTLWE2TRLWE__<<<1, NUM_THREAD4HOMGATE,
                            MEM4HOMGATE, st>>>
        (out, in, mu, bk_ntts[gpuNum], *ntt_handlers[gpuNum]);
    CuCheckError();
}

void SEIandBootstrap2TRLWE(TFHEpp::lvl1param::T* out, TFHEpp::lvl1param::T* in,
                         lvl1param::T mu, hipStream_t st, const int gpuNum)
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        __SEIandBootstrap2TRLWE__), hipFuncAttributeMaxDynamicSharedMemorySize,
        ((2*lvl1param::l + 3) * lvl1param::n + (lvl0param::n + 1)/2+1) * sizeof(FFP));
    __SEIandBootstrap2TRLWE__<<<1, lvl1param::l * lvl1param::n>>
                                NTT_THRED_UNITBIT,
                            ((2*lvl1param::l + 3) * lvl1param::n + (lvl0param::n + 1)/2+1) * sizeof(FFP), st>>>
        (out, in, mu, bk_ntts[gpuNum], ksk_devs[gpuNum], *ntt_handlers[gpuNum]);
    CuCheckError();
}

void NandBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                   TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        __NandBootstrap__), hipFuncAttributeMaxDynamicSharedMemorySize,
        MEM4HOMGATE);
    __NandBootstrap__<<<1, NUM_THREAD4HOMGATE, MEM4HOMGATE,
                      st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void OrBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                 TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        __OrBootstrap__), hipFuncAttributeMaxDynamicSharedMemorySize,
        MEM4HOMGATE);
    __OrBootstrap__<<<1, NUM_THREAD4HOMGATE, MEM4HOMGATE,
                    st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void OrYNBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                   TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        __OrYNBootstrap__), hipFuncAttributeMaxDynamicSharedMemorySize,
        MEM4HOMGATE);
    __OrYNBootstrap__<<<1, NUM_THREAD4HOMGATE, MEM4HOMGATE,
                      st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void OrNYBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                   TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        __OrNYBootstrap__), hipFuncAttributeMaxDynamicSharedMemorySize,
        MEM4HOMGATE);
    __OrNYBootstrap__<<<1, NUM_THREAD4HOMGATE, MEM4HOMGATE,
                      st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void AndBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                  TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        __AndBootstrap__), hipFuncAttributeMaxDynamicSharedMemorySize,
        MEM4HOMGATE);
    __AndBootstrap__<<<1, NUM_THREAD4HOMGATE, MEM4HOMGATE,
                     st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void AndYNBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                    TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        __AndYNBootstrap__), hipFuncAttributeMaxDynamicSharedMemorySize,
        MEM4HOMGATE);
    __AndYNBootstrap__<<<1, NUM_THREAD4HOMGATE, MEM4HOMGATE,
                       st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void AndNYBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                    TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        __AndNYBootstrap__), hipFuncAttributeMaxDynamicSharedMemorySize,
        MEM4HOMGATE);
    __AndNYBootstrap__<<<1, NUM_THREAD4HOMGATE, MEM4HOMGATE,
                       st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void NorBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                  TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        __NorBootstrap__), hipFuncAttributeMaxDynamicSharedMemorySize,
        MEM4HOMGATE);
    __NorBootstrap__<<<1, NUM_THREAD4HOMGATE, MEM4HOMGATE,
                     st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void XorBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                  TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        __XorBootstrap__), hipFuncAttributeMaxDynamicSharedMemorySize,
        MEM4HOMGATE);
    __XorBootstrap__<<<1, NUM_THREAD4HOMGATE, MEM4HOMGATE,
                     st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void XnorBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in0,
                   TFHEpp::lvl0param::T* in1, hipStream_t st, const int gpuNum)
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        __XnorBootstrap__), hipFuncAttributeMaxDynamicSharedMemorySize,
        MEM4HOMGATE);
    __XnorBootstrap__<<<1, NUM_THREAD4HOMGATE, MEM4HOMGATE,
                      st>>>
        (out, in0, in1, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void CopyBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in,
                   hipStream_t st, const int gpuNum)
{
    __CopyBootstrap__<<<1, lvl0param::n + 1, 0, st>>>(out, in);
    CuCheckError();
}

void NotBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* in,
                  hipStream_t st, const int gpuNum)
{
    __NotBootstrap__<<<1, lvl0param::n + 1, 0, st>>>(out, in);
    CuCheckError();
}

void MuxBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* inc,
                  TFHEpp::lvl0param::T* in1, TFHEpp::lvl0param::T* in0,
                  hipStream_t st, const int gpuNum)
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        __MuxBootstrap__), hipFuncAttributeMaxDynamicSharedMemorySize,
        (2 * lvl1param::l + 3) * lvl1param::n * sizeof(FFP));
    __MuxBootstrap__<<<1, NUM_THREAD4HOMGATE,
                     (2 * lvl1param::l + 3) * lvl1param::n * sizeof(FFP),
                     st>>>
        (out, inc, in1, in0, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}

void NMuxBootstrap(TFHEpp::lvl0param::T* out, TFHEpp::lvl0param::T* inc,
                  TFHEpp::lvl0param::T* in1, TFHEpp::lvl0param::T* in0,
                  hipStream_t st, const int gpuNum)
{
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        __NMuxBootstrap__), hipFuncAttributeMaxDynamicSharedMemorySize,
        (2 * lvl1param::l + 3) * lvl1param::n * sizeof(FFP));
    __NMuxBootstrap__<<<1, NUM_THREAD4HOMGATE,
                     (2 * lvl1param::l + 3) * lvl1param::n * sizeof(FFP),
                     st>>>
        (out, inc, in1, in0, bk_ntts[gpuNum], ksk_devs[gpuNum],
         *ntt_handlers[gpuNum]);
    CuCheckError();
}
}  // namespace cufhe
