/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <unistd.h>

#include <array>
#include <cloudkey.hpp>
#include <include/bootstrap_gpu.cuh>
#include <include/keyswitch_gpu.cuh>
#include <include/cufhe_gpu.cuh>
#include <params.hpp>

namespace cufhe {

int _gpuNum = 1;

int streamCount = 0;

void SetGPUNum(int gpuNum) { _gpuNum = gpuNum; }

void Initialize() { InitializeNTThandlers(_gpuNum); }

void Initialize(const TFHEpp::EvalKey& ek)
{
    InitializeNTThandlers(_gpuNum);
    BootstrappingKeyToNTT(*ek.bklvl01, _gpuNum);
    KeySwitchingKeyToDevice(*ek.iksklvl10, _gpuNum);
}

void CleanUp()
{
    DeleteBootstrappingKeyNTT(_gpuNum);
    DeleteKeySwitchingKey(_gpuNum);
}

void CMUXNTT(cuFHETRLWElvl1& res, cuFHETRGSWNTTlvl1& cs, cuFHETRLWElvl1& c1,
             cuFHETRLWElvl1& c0, Stream st)
{
    hipSetDevice(st.device_id());
    hipMemcpyAsync(cs.trgswdevices[st.device_id()], cs.trgswhost.data(),
                    sizeof(cs.trgswhost), hipMemcpyHostToDevice, st.st());
    hipMemcpyAsync(c1.trlwedevices[st.device_id()], c1.trlwehost.data(),
                    sizeof(c1.trlwehost), hipMemcpyHostToDevice, st.st());
    hipMemcpyAsync(c0.trlwedevices[st.device_id()], c0.trlwehost.data(),
                    sizeof(c0.trlwehost), hipMemcpyHostToDevice, st.st());
    CMUXNTTkernel(res.trlwedevices[st.device_id()],
                  cs.trgswdevices[st.device_id()],
                  c1.trlwedevices[st.device_id()],
                  c0.trlwedevices[st.device_id()], st.st(), st.device_id());
    hipMemcpyAsync(res.trlwehost.data(), res.trlwedevices[st.device_id()],
                    sizeof(res.trlwehost), hipMemcpyDeviceToHost, st.st());
}

void GateBootstrappingTLWE2TRLWElvl01NTT(cuFHETRLWElvl1& out, Ctxt<TFHEpp::lvl0param>& in,
                                         Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D<TFHEpp::lvl0param>(in, st);
    BootstrapTLWE2TRLWE(out.trlwedevices[st.device_id()],
                        in.tlwedevices[st.device_id()], TFHEpp::lvl1param::μ,
                        st.st(), st.device_id());
    hipMemcpyAsync(out.trlwehost.data(), out.trlwedevices[st.device_id()],
                    sizeof(out.trlwehost), hipMemcpyDeviceToHost, st.st());
}

void gGateBootstrappingTLWE2TRLWElvl01NTT(cuFHETRLWElvl1& out, Ctxt<TFHEpp::lvl0param>& in,
                                          Stream st)
{
    hipSetDevice(st.device_id());
    BootstrapTLWE2TRLWE(out.trlwedevices[st.device_id()],
                        in.tlwedevices[st.device_id()], TFHEpp::lvl1param::μ,
                        st.st(), st.device_id());
}

void Refresh(cuFHETRLWElvl1& out, cuFHETRLWElvl1& in, Stream st)
{
    hipSetDevice(st.device_id());
    hipMemcpyAsync(in.trlwedevices[st.device_id()], in.trlwehost.data(),
                    sizeof(in.trlwehost), hipMemcpyHostToDevice, st.st());
    SEIandBootstrap2TRLWE(out.trlwedevices[st.device_id()],
                          in.trlwedevices[st.device_id()], TFHEpp::lvl1param::μ,
                          st.st(), st.device_id());
    hipMemcpyAsync(out.trlwehost.data(), out.trlwedevices[st.device_id()],
                    sizeof(out.trlwehost), hipMemcpyDeviceToHost, st.st());
}

void gRefresh(cuFHETRLWElvl1& out, cuFHETRLWElvl1& in, Stream st)
{
    hipSetDevice(st.device_id());
    BootstrapTLWE2TRLWE(out.trlwedevices[st.device_id()],
                        in.trlwedevices[st.device_id()], TFHEpp::lvl1param::μ,
                        st.st(), st.device_id());
}

void SampleExtractAndKeySwitch(Ctxt<TFHEpp::lvl0param>& out, const cuFHETRLWElvl1& in, Stream st)
{
    hipSetDevice(st.device_id());
    hipMemcpyAsync(in.trlwedevices[st.device_id()], in.trlwehost.data(),
                    sizeof(in.trlwehost), hipMemcpyHostToDevice, st.st());
    SEIandKS(out.tlwedevices[st.device_id()], in.trlwedevices[st.device_id()],
            st.st(), st.device_id());
    CtxtCopyD2H<TFHEpp::lvl0param>(out, st);
}

void gSampleExtractAndKeySwitch(Ctxt<TFHEpp::lvl0param>& out, const cuFHETRLWElvl1& in, Stream st)
{
    hipSetDevice(st.device_id());
    hipMemcpyAsync(in.trlwedevices[st.device_id()], in.trlwehost.data(),
                    sizeof(in.trlwehost), hipMemcpyHostToDevice, st.st());
    SEIandKS(out.tlwedevices[st.device_id()], in.trlwedevices[st.device_id()],
            st.st(), st.device_id());
}

void Nand(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D<TFHEpp::lvl0param>(in0, st);
    CtxtCopyH2D<TFHEpp::lvl0param>(in1, st);
    NandBootstrap<TFHEpp::lvl01param, TFHEpp::lvl1param::μ, TFHEpp::lvl10param>(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H<TFHEpp::lvl0param>(out, st);
}

void gNand(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    NandBootstrap<TFHEpp::lvl01param, TFHEpp::lvl1param::μ, TFHEpp::lvl10param>(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void Or(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D<TFHEpp::lvl0param>(in0, st);
    CtxtCopyH2D<TFHEpp::lvl0param>(in1, st);
    OrBootstrap(out.tlwedevices[st.device_id()],
                in0.tlwedevices[st.device_id()],
                in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H<TFHEpp::lvl0param>(out, st);
}

void gOr(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    OrBootstrap(out.tlwedevices[st.device_id()],
                in0.tlwedevices[st.device_id()],
                in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void OrYN(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D<TFHEpp::lvl0param>(in0, st);
    CtxtCopyH2D<TFHEpp::lvl0param>(in1, st);
    OrYNBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H<TFHEpp::lvl0param>(out, st);
}

void gOrYN(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    OrYNBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void OrNY(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D<TFHEpp::lvl0param>(in0, st);
    CtxtCopyH2D<TFHEpp::lvl0param>(in1, st);
    OrNYBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H<TFHEpp::lvl0param>(out, st);
}

void gOrNY(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    OrNYBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void And(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D<TFHEpp::lvl0param>(in0, st);
    CtxtCopyH2D<TFHEpp::lvl0param>(in1, st);
    AndBootstrap(out.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H<TFHEpp::lvl0param>(out, st);
}

void gAnd(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    AndBootstrap(out.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void AndYN(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D<TFHEpp::lvl0param>(in0, st);
    CtxtCopyH2D<TFHEpp::lvl0param>(in1, st);
    AndYNBootstrap(out.tlwedevices[st.device_id()],
                   in0.tlwedevices[st.device_id()],
                   in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H<TFHEpp::lvl0param>(out, st);
}

void gAndYN(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    AndYNBootstrap(out.tlwedevices[st.device_id()],
                   in0.tlwedevices[st.device_id()],
                   in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void AndNY(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D<TFHEpp::lvl0param>(in0, st);
    CtxtCopyH2D<TFHEpp::lvl0param>(in1, st);
    AndNYBootstrap(out.tlwedevices[st.device_id()],
                   in0.tlwedevices[st.device_id()],
                   in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H<TFHEpp::lvl0param>(out, st);
}

void gAndNY(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    AndNYBootstrap(out.tlwedevices[st.device_id()],
                   in0.tlwedevices[st.device_id()],
                   in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void Nor(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D<TFHEpp::lvl0param>(in0, st);
    CtxtCopyH2D<TFHEpp::lvl0param>(in1, st);
    NorBootstrap(out.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H<TFHEpp::lvl0param>(out, st);
}

void gNor(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    NorBootstrap(out.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void Xor(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D<TFHEpp::lvl0param>(in0, st);
    CtxtCopyH2D<TFHEpp::lvl0param>(in1, st);
    XorBootstrap(out.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H<TFHEpp::lvl0param>(out, st);
}

void gXor(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    XorBootstrap(out.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void Xnor(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D<TFHEpp::lvl0param>(in0, st);
    CtxtCopyH2D<TFHEpp::lvl0param>(in1, st);
    XnorBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H<TFHEpp::lvl0param>(out, st);
}

void gXnor(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in0, Ctxt<TFHEpp::lvl0param>& in1, Stream st)
{
    hipSetDevice(st.device_id());
    XnorBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void Not(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D<TFHEpp::lvl0param>(in, st);
    NotBootstrap(out.tlwedevices[st.device_id()],
                 in.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H<TFHEpp::lvl0param>(out, st);
}

void gNot(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in, Stream st)
{
    hipSetDevice(st.device_id());
    NotBootstrap(out.tlwedevices[st.device_id()],
                 in.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void Copy(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D<TFHEpp::lvl0param>(in, st);
    CopyBootstrap(out.tlwedevices[st.device_id()],
                  in.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H<TFHEpp::lvl0param>(out, st);
}

void gCopy(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in, Stream st)
{
    hipSetDevice(st.device_id());
    CopyBootstrap(out.tlwedevices[st.device_id()],
                  in.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void CopyOnHost(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& in) { out.tlwehost = in.tlwehost; }

// Mux(inc,in1,in0) = inc?in1:in0 = inc&in1 + (!inc)&in0
void Mux(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& inc, Ctxt<TFHEpp::lvl0param>& in1, Ctxt<TFHEpp::lvl0param>& in0, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D<TFHEpp::lvl0param>(inc, st);
    CtxtCopyH2D<TFHEpp::lvl0param>(in1, st);
    CtxtCopyH2D<TFHEpp::lvl0param>(in0, st);
    MuxBootstrap(out.tlwedevices[st.device_id()],
                 inc.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H<TFHEpp::lvl0param>(out, st);
}

void gMux(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& inc, Ctxt<TFHEpp::lvl0param>& in1, Ctxt<TFHEpp::lvl0param>& in0, Stream st)
{
    hipSetDevice(st.device_id());
    MuxBootstrap(out.tlwedevices[st.device_id()],
                 inc.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()], st.st(), st.device_id());
}

void NMux(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& inc, Ctxt<TFHEpp::lvl0param>& in1, Ctxt<TFHEpp::lvl0param>& in0, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D<TFHEpp::lvl0param>(inc, st);
    CtxtCopyH2D<TFHEpp::lvl0param>(in1, st);
    CtxtCopyH2D<TFHEpp::lvl0param>(in0, st);
    NMuxBootstrap(out.tlwedevices[st.device_id()],
                  inc.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H<TFHEpp::lvl0param>(out, st);
}

void gNMux(Ctxt<TFHEpp::lvl0param>& out, Ctxt<TFHEpp::lvl0param>& inc, Ctxt<TFHEpp::lvl0param>& in1, Ctxt<TFHEpp::lvl0param>& in0, Stream st)
{
    hipSetDevice(st.device_id());
    NMuxBootstrap(out.tlwedevices[st.device_id()],
                  inc.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()], st.st(), st.device_id());
}

bool StreamQuery(Stream st)
{
    hipSetDevice(st.device_id());
    hipError_t res = hipStreamQuery(st.st());
    if (res == hipSuccess) {
        return true;
    }
    else {
        return false;
    }
}
}  // namespace cufhe
