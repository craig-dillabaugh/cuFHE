/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <include/cufhe.h>
#include <unistd.h>

#include <array>
#include <include/bootstrap_gpu.cuh>
#include <include/cufhe_gpu.cuh>
#include "../thirdparties/TFHEpp/include/cloudkey.hpp"
#include "../thirdparties/TFHEpp/include/params.hpp"

namespace cufhe {

int _gpuNum = 1;

int streamCount = 0;

void SetGPUNum(int gpuNum) { _gpuNum = gpuNum; }

void Initialize(const TFHEpp::GateKeywoFFT& gk)
{
    BootstrappingKeyToNTT(gk.bklvl01, _gpuNum);
    KeySwitchingKeyToDevice(gk.ksk, _gpuNum);
}

void CleanUp()
{
    DeleteBootstrappingKeyNTT(_gpuNum);
    DeleteKeySwitchingKey(_gpuNum);
}

inline void CtxtCopyH2D(Ctxt& c, Stream st)
{
    hipSetDevice(st.device_id());
    hipMemcpyAsync(c.tlwedevices[st.device_id()],
                    c.tlwehost.data(), sizeof(c.tlwehost),
                    hipMemcpyHostToDevice, st.st());
}

inline void CtxtCopyD2H(Ctxt& c, Stream st)
{
    hipSetDevice(st.device_id());
    hipMemcpyAsync(c.tlwehost.data(),
                    c.tlwedevices[st.device_id()],
                    sizeof(c.tlwehost), hipMemcpyDeviceToHost, st.st());
}

void GateBootstrappingTLWE2TRLWElvl01NTT(cuFHETRLWElvl1& out, Ctxt& in,
                                         Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in, st);
    BootstrapTLWE2TRLWE(out.trlwedevices[st.device_id()],
                        in.tlwedevices[st.device_id()], 1U<<29, st.st(),
                        st.device_id());
    hipMemcpyAsync(out.trlwehost.data(), out.trlwedevices[st.device_id()],
                    sizeof(out.trlwehost), hipMemcpyDeviceToHost, st.st());
}

void gGateBootstrappingTLWE2TRLWElvl01NTT(cuFHETRLWElvl1& out, Ctxt& in,
                                          Stream st)
{
    hipSetDevice(st.device_id());
    BootstrapTLWE2TRLWE(out.trlwedevices[st.device_id()],
                        in.tlwedevices[st.device_id()], 1U<<29, st.st(),
                        st.device_id());
    hipMemcpyAsync(out.trlwehost.data(), out.trlwedevices[st.device_id()],
                    sizeof(out.trlwehost), hipMemcpyDeviceToHost, st.st());
}

void SampleExtractAndKeySwitch(Ctxt& out, const cuFHETRLWElvl1& in, Stream st)
{
    hipSetDevice(st.device_id());
    hipMemcpyAsync(in.trlwedevices[st.device_id()], in.trlwehost.data(),
                    sizeof(in.trlwehost), hipMemcpyHostToDevice, st.st());
    SEandKS(out.tlwedevices[st.device_id()],
            in.trlwedevices[st.device_id()], st.st(), st.device_id());
    CtxtCopyD2H(out, st);
}

void gSampleExtractAndKeySwitch(Ctxt& out, const cuFHETRLWElvl1& in, Stream st)
{
    hipSetDevice(st.device_id());
    hipMemcpyAsync(in.trlwedevices[st.device_id()], in.trlwehost.data(),
                    sizeof(in.trlwehost), hipMemcpyHostToDevice, st.st());
    SEandKS(out.tlwedevices[st.device_id()],
            in.trlwedevices[st.device_id()], st.st(), st.device_id());
}

void Nand(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    NandBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(),
                  st.device_id());
    CtxtCopyD2H(out, st);
}

void gNand(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    NandBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(),
                  st.device_id());
}

void Or(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    OrBootstrap(out.tlwedevices[st.device_id()],
                in0.tlwedevices[st.device_id()],
                in1.tlwedevices[st.device_id()], st.st(),
                st.device_id());
    CtxtCopyD2H(out, st);
}

void gOr(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    OrBootstrap(out.tlwedevices[st.device_id()],
                in0.tlwedevices[st.device_id()],
                in1.tlwedevices[st.device_id()], st.st(),
                st.device_id());
}

void OrYN(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    OrYNBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(),
                  st.device_id());
    CtxtCopyD2H(out, st);
}

void gOrYN(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    OrYNBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(),
                  st.device_id());
}

void OrNY(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    OrNYBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(),
                  st.device_id());
    CtxtCopyD2H(out, st);
}

void gOrNY(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    OrNYBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(),
                  st.device_id());
}

void And(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    AndBootstrap(out.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()], st.st(),
                 st.device_id());
    CtxtCopyD2H(out, st);
}

void gAnd(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    AndBootstrap(out.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()], st.st(),
                 st.device_id());
}

void AndYN(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    AndYNBootstrap(out.tlwedevices[st.device_id()],
                   in0.tlwedevices[st.device_id()],
                   in1.tlwedevices[st.device_id()], st.st(),
                   st.device_id());
    CtxtCopyD2H(out, st);
}

void gAndYN(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    AndYNBootstrap(out.tlwedevices[st.device_id()],
                   in0.tlwedevices[st.device_id()],
                   in1.tlwedevices[st.device_id()], st.st(),
                   st.device_id());
}

void AndNY(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    AndNYBootstrap(out.tlwedevices[st.device_id()],
                   in0.tlwedevices[st.device_id()],
                   in1.tlwedevices[st.device_id()], st.st(),
                   st.device_id());
    CtxtCopyD2H(out, st);
}

void gAndNY(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    AndNYBootstrap(out.tlwedevices[st.device_id()],
                   in0.tlwedevices[st.device_id()],
                   in1.tlwedevices[st.device_id()], st.st(),
                   st.device_id());
}

void Nor(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    NorBootstrap(out.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()], st.st(),
                 st.device_id());
    CtxtCopyD2H(out, st);
}

void gNor(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    NorBootstrap(out.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()], st.st(),
                 st.device_id());
}

void Xor(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    XorBootstrap(out.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()], st.st(),
                 st.device_id());
    CtxtCopyD2H(out, st);
}

void gXor(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    XorBootstrap(out.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()], st.st(),
                 st.device_id());
}

void Xnor(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in0, st);
    CtxtCopyH2D(in1, st);
    XnorBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(),
                  st.device_id());
    CtxtCopyD2H(out, st);
}

void gXnor(Ctxt& out, Ctxt& in0, Ctxt& in1, Stream st)
{
    hipSetDevice(st.device_id());
    XnorBootstrap(out.tlwedevices[st.device_id()],
                  in0.tlwedevices[st.device_id()],
                  in1.tlwedevices[st.device_id()], st.st(),
                  st.device_id());
}

void Not(Ctxt& out, Ctxt& in, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in, st);
    NotBootstrap(out.tlwedevices[st.device_id()],
                 in.tlwedevices[st.device_id()], st.st(),
                 st.device_id());
    CtxtCopyD2H(out, st);
}

void gNot(Ctxt& out, Ctxt& in, Stream st)
{
    hipSetDevice(st.device_id());
    NotBootstrap(out.tlwedevices[st.device_id()],
                 in.tlwedevices[st.device_id()], st.st(),
                 st.device_id());
}

void Copy(Ctxt& out, Ctxt& in, Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(in, st);
    CopyBootstrap(out.tlwedevices[st.device_id()],
                  in.tlwedevices[st.device_id()], st.st(),
                  st.device_id());
    CtxtCopyD2H(out, st);
}

void gCopy(Ctxt& out, Ctxt& in, Stream st)
{
    hipSetDevice(st.device_id());
    CopyBootstrap(out.tlwedevices[st.device_id()],
                  in.tlwedevices[st.device_id()], st.st(),
                  st.device_id());
}

void CopyOnHost(Ctxt& out, Ctxt& in)
{
    out.tlwehost = in.tlwehost;
}

// Mux(inc,in1,in0) = inc?in1:in0 = inc&in1 + (!inc)&in0
void Mux(Ctxt& out, Ctxt& inc, Ctxt& in1, Ctxt& in0,
         Stream st)
{
    hipSetDevice(st.device_id());
    CtxtCopyH2D(inc, st);
    CtxtCopyH2D(in1, st);
    CtxtCopyH2D(in0, st);
    MuxBootstrap(out.tlwedevices[st.device_id()],
                 inc.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()], st.st(),
                 st.device_id());
    CtxtCopyD2H(out, st);
}

void gMux(Ctxt& out, Ctxt& inc, Ctxt& in1, Ctxt& in0,
          Stream st)
{
    hipSetDevice(st.device_id());
    MuxBootstrap(out.tlwedevices[st.device_id()],
                 inc.tlwedevices[st.device_id()],
                 in1.tlwedevices[st.device_id()],
                 in0.tlwedevices[st.device_id()], st.st(),
                 st.device_id());
}

// void SetToGPU(Ctxt& in)
// {
//     hipMemcpy(in.lwe_sample_device_->data(), in.lwe_sample_->data(),
//                in.lwe_sample_->SizeData(), hipMemcpyHostToDevice);
// }

// void GetFromGPU(Ctxt& out)
// {
//     hipMemcpy(out.lwe_sample_->data(), out.lwe_sample_device_->data(),
//                out.lwe_sample_->SizeData(), hipMemcpyDeviceToHost);
// }

bool StreamQuery(Stream st)
{
    hipSetDevice(st.device_id());
    hipError_t res = hipStreamQuery(st.st());
    if (res == hipSuccess) {
        return true;
    }
    else {
        return false;
    }
}
}  // namespace cufhe
