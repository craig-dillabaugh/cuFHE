#include "hip/hip_runtime.h"
#include <include/keyswitch_gpu.cuh>
namespace cufhe{

vector<lvl0param::T*> ksk_devs;

void KeySwitchingKeyToDevice(const KeySwitchingKey<lvl10param>& ksk,
                             const int gpuNum)
{
    ksk_devs.resize(gpuNum);
    for (int i = 0; i < gpuNum; i++) {
        hipSetDevice(i);
        hipMalloc((void**)&ksk_devs[i], sizeof(ksk));
        CuSafeCall(hipMemcpy(ksk_devs[i], ksk.data(), sizeof(ksk),
                              hipMemcpyHostToDevice));
    }
}

void DeleteKeySwitchingKey(const int gpuNum)
{
    for (int i = 0; i < ksk_devs.size(); i++) {
        hipSetDevice(i);
        hipFree(ksk_devs[i]);
    }
}

template<class P>
__global__ void __SEIandKS__(typename P::targetP::T* const out, const typename P::domainP::T* const in,
                            const typename P::targetP::T* const ksk)
{
    KeySwitch<lvl10param>(out, in, ksk);
    __threadfence();
}

void SEIandKS(TFHEpp::lvl0param::T* const out, const TFHEpp::lvl1param::T* const in,
             const hipStream_t& st, const int gpuNum)
{
    __SEIandKS__<TFHEpp::lvl10param><<<1, lvl0param::n + 1, 0, st>>>(out, in,
                                                ksk_devs[gpuNum]);
    CuCheckError();
}
}